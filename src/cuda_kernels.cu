#include "hip/hip_runtime.h"
#include "../include/cuda_kernels.hpp"

/*
 * Class constructor to retrieve all relevant cuda variables. May be removed later, but figured it might be
 * useful to have this information when deciding on block or thread counts.
 */

cudaKernel::cudaKernel() {
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, 0);

    streamingProcs = devProp.multiProcessorCount;
    maxThreadsPerBlock = devProp.maxThreadsPerBlock;
    totalMem = devProp.totalGlobalMem;
    sharedMemPerBlock = devProp.sharedMemPerBlock;
    regPerBlock = devProp.regsPerBlock;
    concurrentCopy = (devProp.deviceOverlap ? 1 : 0);
    name = devProp.name;
}

/*
 * Checks to see if an error has occcured
 */

void check_error(hipError_t status, const char *msg)
{
    if (status != hipSuccess)
    {
        const char *errorStr = hipGetErrorString(status);
        printf("%s:\n%s\nError Code: %d\n\n", msg, errorStr, status);
        exit(status); // bail out immediately (makes debugging easier)
    }
}

/*
 * Function is passed the image as represented by an array of unsigned chars. Data is then written into the second
 * parameter and then retrieved by caller.
 */

__global__ void cuda_gaussian_blur(const uchar *image, uchar *returnImage, const uint64 length, int kernelSize,
        float * conv, const uint64 rows, const uint64 cols) {

    int global_id = 3.0 * (blockIdx.x * blockDim.x + threadIdx.x); // have pixel working on
    double b = 0.0, g = 0.0, r = 0.0;

    if(global_id < length) { // we have not exceeded the range of our array
        for (int y1 = -kernelSize / 2; y1 <= kernelSize / 2; y1++) { // loop through y val of conv matrix
            for (int x1 = -kernelSize / 2; x1 <= kernelSize / 2; x1++) { // loop through x val of conv matrix
                uint64 x = ((global_id/3)%cols + x1), y = ((global_id/3)/rows + y1); // have master x and y coord
                uint64 temp_id = y*rows + x; // now have 2D to 1D conversion
                if (y + y1 >= 0 && y + y1 < rows) { // check to see if out of bounds of rows
                    if (x + x1 >= 0 && x + x1 < cols) {
                        b += image[temp_id] * conv[(kernelSize / 2 + y1) * kernelSize + (kernelSize / 2 + x1)];     // B
                        g += image[temp_id + 1] * conv[(kernelSize / 2 + y1) * kernelSize + (kernelSize / 2 + x1)]; // G
                        r += image[temp_id + 2] * conv[(kernelSize / 2 + y1) * kernelSize + (kernelSize / 2 + x1)]; // R
                    }
                }
            }
        }
        returnImage[global_id] = b;
        returnImage[global_id + 1] = g;
        returnImage[global_id + 2] = r;
    }
}

/*
 * Function takes CV Mat, and intensity and sets up the call so that a CUDA device can accelerate the task.
 * Function then cleans up and returns the result.
 */

cv::Mat cudaKernel::gaussian_blur(const cv::Mat &frame, int kernelSize, float sigma) {
    cv::Mat finalResult; // return data
    hipError_t statusA; // error of initialization
    uchar *dev_imageA; // gpu pointer for image
    uchar *dev_imageB; // gpu pointer to return image
    std::vector<uchar> *array = new std::vector<uchar>; // frame into vector
    std::vector<uchar> *results; // vector into frame
    uchar *host_image; // array for frame
    float *conv = (float*)malloc(kernelSize*kernelSize*sizeof(float));
    float *dev_conv;

    if (frame.isContinuous()) {
        array->assign(frame.data, frame.data + frame.total());
    } else {
        for (int i = 0; i < frame.rows; ++i) {
            array->insert(array->end(), frame.ptr<uchar>(i), frame.ptr<uchar>(i) + frame.cols);
        }
    }

    if(kernelSize > 1 && kernelSize%2) {
        helper::gaussian_convolution(conv, kernelSize, sigma);
    } else if(kernelSize == 1 && kernelSize%2) {
        conv[0] = 1;
    } else {
        delete(conv);
        throw "Kernel size cannot be less than one and kernel size must be odd";
    }

    host_image = array->data(); // convert image vector to prep array to copy to GPU
    statusA = hipMalloc(&dev_imageA, array->size()*sizeof(uchar)); // allocate mem on gpu
    check_error(statusA, "Failed Allocation 1");
    statusA = hipMalloc(&dev_imageB, array->size()* sizeof(uchar)); // allocate mem on gpu
    check_error(statusA, "Failed Allocation 2");
    statusA = hipMalloc(&dev_conv, kernelSize*kernelSize* sizeof(float));
    check_error(statusA, "Failed Allocation 3");

    statusA = hipMemcpy(dev_imageA, host_image, array->size()*sizeof(uchar), hipMemcpyHostToDevice);
    check_error(statusA, "Failed cuda host to device copy 1");
    statusA = hipMemcpy(dev_conv, conv, kernelSize*kernelSize*sizeof(float), hipMemcpyHostToDevice);
    check_error(statusA, "Failed cuda host to device copy 2");

    uint32_t numBlocks = (int)ceilf((float)(frame.rows*frame.cols)/1024);

    cuda_gaussian_blur<<<numBlocks, 1024>>>(dev_imageA, dev_imageB, array->size()*sizeof(uchar), kernelSize, dev_conv, frame.rows, frame.cols);
    check_error(hipGetLastError(), "Error in kernel.");

    statusA = hipMemcpy(host_image, dev_imageA, array->size()* sizeof(uchar), hipMemcpyDeviceToHost);
    check_error(statusA, "Failed cuda device to host copy");

    results = new std::vector<uchar>(host_image, host_image+array->size());
    finalResult = cv::Mat(frame.rows, frame.cols, frame.type(), results->data());

    statusA = hipFree(dev_imageA);
    check_error(statusA, "Failed to free device memory 1");
    statusA = hipFree(dev_imageB);
    check_error(statusA, "Failed to free device memory 2");

    delete(results);
    delete(array);
    delete(conv);

    return finalResult;
}